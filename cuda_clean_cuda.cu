#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

	// определение параметров GPU
#define THREADS 128
#define BLOCKS 32
#define GENS 32*128

	// определение входных параметров
#define L 200
#define PN 2000
#define SEED 87654321
#define TN 1000
#define P 10


	// проверка ошибок CUDA
#define cudaCheck(cudaf) { cudaCheckInner((cudaf), __FILE__, __LINE__); }
int cudaCheckInner(hipError_t code, const char* file, int line) {

	if (code != hipSuccess) {

		fprintf(stderr, "CUDA failed: %s %s %d\n", hipGetErrorString(code), file, line);
		return 1;

	}
	else return 0;

}


	// структура узел
struct node {
	unsigned int SH;
	unsigned int SV;
};

	// структура частица 
struct particle {
	int x;
	int y;
};


	// инициализация генератора
__global__ void initfGENS(hiprandStatePhilox4_32_10_t* d_gen) {

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(SEED, id, 0, d_gen + id);

}


	// инициализация решетки
__global__ void initfGRID(struct node* d_grid, hiprandStatePhilox4_32_10_t* d_gen) {

	int id = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = id; i < L * L; i += GENS) {

		d_grid[i].SH = (hiprand(&d_gen [id]) % 10) < P;
		d_grid[i].SV = (hiprand(&d_gen [id]) % 10) < P;

	}

}


	// инициализация частиц
__global__ void initfDOT(struct particle* d_dot, hiprandStatePhilox4_32_10_t* d_gen) {

	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < PN; id += GENS) {

		d_dot[id].x = L / 2;
		d_dot[id].y = L / 2;

	}

}


	// шаг
__global__ void step(struct node* d_grid, struct particle* d_dot, int* d_exc, hiprandStatePhilox4_32_10_t* d_gen) {

	unsigned int r;
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = id; i < PN; i += GENS) {

		int xy = d_dot [i].y * L + d_dot [i].x;

			// 1
		if ((d_grid[xy].SH == 1) && (d_grid[xy - 1].SH == 0) && (d_grid [xy].SV == 0) && (d_grid [xy - L].SV == 0)) {
			d_dot[i].x++;
		}
			// 2
		if ((d_grid [xy].SH == 0) && (d_grid [xy - 1].SH == 1) && (d_grid [xy].SV == 0) && (d_grid [xy - L].SV == 0)) {
			d_dot[i].x--;
		}
			// 3
		if ((d_grid [xy].SH == 0) && (d_grid [xy - 1].SH == 0) && (d_grid [xy].SV == 1) && (d_grid [xy - L].SV == 0)) {
			d_dot[i].y++;
		}
			 // 4
		if ((d_grid [xy].SH == 0) && (d_grid [xy - 1].SH == 0) && (d_grid [xy].SV == 0) && (d_grid [xy - L].SV == 1)) {
			d_dot[i].y--;
		}
					// 12
				if ((d_grid [xy].SH == 1) && (d_grid [xy - 1].SH == 1) && (d_grid [xy].SV == 0) && (d_grid [xy - L].SV == 0)) {
					r = hiprand(&d_gen [id]) % 2;
					if (r == 0) d_dot[i].x++;
					else d_dot[i].x--;
				}
					// 13
				if ((d_grid [xy].SH == 1) && (d_grid [xy - 1].SH == 0) && (d_grid [xy].SV == 1) && (d_grid [xy - L].SV == 0)) {
					r = hiprand(&d_gen [id]) % 2;
					if (r == 0) d_dot[i].x++;
					else d_dot[i].y++;
				}
					// 14
				if ((d_grid [xy].SH == 1) && (d_grid [xy - 1].SH == 0) && (d_grid [xy].SV == 0) && (d_grid [xy - L].SV == 1)) {
					r = hiprand(&d_gen [id]) % 2;
					if (r == 0) d_dot[i].x++;
					else d_dot[i].y--;
				}
					// 23
				if ((d_grid [xy].SH == 0) && (d_grid [xy - 1].SH == 1) && (d_grid [xy].SV == 1) && (d_grid [xy - L].SV == 0)) {
					r = hiprand(&d_gen [id]) % 2;
					if (r == 0) d_dot[i].x--;
					else d_dot[i].y++;
				}
					// 24
				if ((d_grid [xy].SH == 0) && (d_grid [xy - 1].SH == 1) && (d_grid [xy].SV == 0) && (d_grid [xy - L].SV == 1)) {
					r = hiprand(&d_gen [id]) % 2;
					if (r == 0) d_dot[i].x--;
					else d_dot[i].y--;
				}
					// 34
				if ((d_grid [xy].SH == 0) && (d_grid [xy - 1].SH == 0) && (d_grid [xy].SV == 1) && (d_grid [xy - L].SV == 1)) {
					r = hiprand(&d_gen [id]) % 2;
					if (r == 0) d_dot[i].y++;
					else d_dot[i].y--;
				}
			// 123
		if ((d_grid [xy].SH == 1) && (d_grid [xy - 1].SH == 1) && (d_grid [xy].SV == 1) && (d_grid [xy - L].SV == 0)) {
			r = hiprand(&d_gen [id]) % 3;
			if (r == 0) d_dot[i].x++;
			else if (r == 1) d_dot[i].x--;
			else d_dot[i].y++;
		}
			// 124
		if ((d_grid [xy].SH == 1) && (d_grid [xy - 1].SH == 1) && (d_grid [xy].SV == 0) && (d_grid [xy - L].SV == 1)) {
			r = hiprand(&d_gen [id]) % 3;
			if (r == 0) d_dot[i].x++;
			else if (r == 1) d_dot[i].x--;
			else d_dot[i].y--;
		}
			// 134
		if ((d_grid [xy].SH == 1) && (d_grid [xy - 1].SH == 0) && (d_grid [xy].SV == 1) && (d_grid [xy - L].SV == 1)) {
			r = hiprand(&d_gen [id]) % 3;
			if (r == 0) d_dot[i].x++;
			else if (r == 1) d_dot[i].y++;
			else d_dot[i].y--;
		}
			// 234
		if ((d_grid [xy].SH == 0) && (d_grid [xy - 1].SH == 1) && (d_grid [xy].SV == 1) && (d_grid [xy - L].SV == 1)) {
			r = hiprand(&d_gen [id]) % 3;
			if (r == 0) d_dot[i].x--;
			else if (r == 1) d_dot[i].y++;
			else d_dot[i].y--;
		}
		
					// 1234
				if ((d_grid [xy].SH == 1) && (d_grid [xy - 1].SH == 1) && (d_grid [xy].SV == 1) && (d_grid [xy - L].SV == 1)) {
					r = hiprand(&d_gen [id]) % 4;
					if (r == 0) d_dot[i].x++;
					else if (r == 1) d_dot[i].x--;
					else if (r == 2) d_dot[i].y++;
					else d_dot[i].y--;
				}

			// допустимость отклонения
		atomicMax(&d_exc[0], max(abs(d_dot[i].x - L / 2), abs(d_dot[i].y - L / 2)));

	}

}


	// вычисление квадрата отклонения
__global__ void rr(float* d_rsq, struct particle* d_dot) {

	for (int id = threadIdx.x + blockIdx.x * blockDim.x; id < PN; id += GENS) {
		atomicAdd(&d_rsq[0], (d_dot[id].x - L / 2) * (d_dot[id].x - L / 2) + (d_dot[id].y - L / 2) * (d_dot[id].y - L / 2));
	}

}


hiprandStatePhilox4_32_10_t* d_gen;
struct node* d_grid;
struct particle* d_dot;

int T = 0;
float rsq [1] = {0};
float* d_rsq;
int exc[1] = {0};
int* d_exc;

int main() {

		// инициализация генератора
	cudaCheck(hipMalloc((void**) &d_gen, GENS * sizeof(hiprandStatePhilox4_32_10_t)));
	initfGENS << < BLOCKS, THREADS >> > (d_gen);
	cudaCheck(hipGetLastError());

		// инициализация решетки
	cudaCheck(hipMalloc((void**) &d_grid, L * L * sizeof(struct node)));
	initfGRID << < BLOCKS, THREADS >> > (d_grid, d_gen);
	cudaCheck(hipGetLastError());

		// инициализация частиц
	cudaCheck(hipMalloc((void**) &d_dot, PN * sizeof(struct particle)));
	initfDOT << < BLOCKS, THREADS >> > (d_dot, d_gen);
	cudaCheck(hipGetLastError());

		// шаги
	cudaCheck(hipMalloc((void**) &d_rsq, 1 * sizeof(float)));
	cudaCheck(hipMemset(d_rsq, 0, 1 * sizeof(float)));
	cudaCheck(hipMalloc((void**) &d_exc, 1 * sizeof(int)));
	cudaCheck(hipMemset(d_exc, 0, 1 * sizeof(int)));

	FILE* out = fopen("output0.txt", "w");
	if (out != NULL) {

		while (T < TN) {

				// шаг
			step << < BLOCKS, THREADS >> > (d_grid, d_dot, d_exc, d_gen);
			cudaCheck(hipGetLastError());
			cudaCheck(hipMemcpy(exc, d_exc, 1 * sizeof(int), hipMemcpyDeviceToHost));

				// запись кратного / "приошибочного"
			if ((T % 10 == 0) || (exc[0] > L / 2)) {

				cudaCheck(hipMemset(d_rsq, 0, 1 * sizeof(float)));
				rr << <BLOCKS, THREADS >> > (d_rsq, d_dot);
				cudaCheck(hipGetLastError());
				cudaCheck(hipMemcpy(rsq, d_rsq, 1 * sizeof(float), hipMemcpyDeviceToHost));
				rsq[0] = rsq[0] / PN;

				fprintf(out, "%f", rsq[0]); fprintf(out, " ");
			}

				// проверка на выход за границу
			if (exc[0] > L / 2) {
				break;
			}

			T++;

		}

		fclose(out);
	}
	else printf("Не удалось открыть файл");

		// освобождение памяти 
	cudaCheck(hipFree(d_gen));
	cudaCheck(hipFree(d_grid));
	cudaCheck(hipFree(d_dot));
	cudaCheck(hipFree(d_rsq));
	cudaCheck(hipFree(d_exc));

	return 0;
}